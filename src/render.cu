#include "hip/hip_runtime.h"


#include <stdio.h>

#include "params.h"


__device__ void d_mult_mat(size_t m1,
                           size_t n1,
                           size_t m2, 
                           size_t n2,
			               const float *a, 
                           const float *b, 
                           float *buf)
{
    for (int i = 0; i < m1; ++i)
        for (int j = 0; j < n2; ++j)
        {
            float sum = 0;
            for (int k = 0; k < m2; ++k)
                sum += a[i * n1 + k] * b[k * n2 + j];
            buf[i * n2 + j] = sum;
        }
}

__device__ void d_rotate_mat(size_t m1,
                             float phi, 
                             char axis,
                             const float *a, 
                             float *buf)
{
	switch (axis)
	{
        case 0:
        {
            float R_x[9] = 
            {
                1,              0,                      0,
                0,              cos(phi),               -1 * sin(phi),
                0,              sin(phi),               cos(phi)
            };
            d_mult_mat(m1, 3, 3, 3, a, R_x, buf);
            break;
        }
        case 1:
        {
            float R_y[9] = 
            {
                cos(phi),                   0,              sin(phi), 
                0,                          1,              0, 
                -1 * sin(phi),              0,              cos(phi)
            };
            d_mult_mat(m1, 3, 3, 3, a, R_y, buf);
            break;
        }
        case 2:
        {
            float R_z[9] = 
            {
                cos(phi),               -1 * sin(phi),              0, 
                sin(phi),               cos(phi),                   0, 
                0,                      0,                          1
            };            
            d_mult_mat(m1, 3, 3, 3, a, R_z, buf);
            break;
        }
	}    
}                             

__device__ void d_trans_mat(size_t m,
                            size_t n, 
                            const float *a, 
                            float *buf)
{
    for (int i = 0; i < m * n; ++i)
    {
        int x = i / m;
        int y = i % m;
        buf[i] = a[n * y + x];
    }
}   

__global__ void render_frame(float A, float B, int offsetx, int offsety, size_t outer, size_t inner, point_t *points)
{

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (col < inner && row < outer)
    {
        int gindex = row * inner + col; //0 .. 28349

        float theta =   (float) row * THETA_INC;
        float phi =     (float) col * PHI_INC;

        //rotate a point of the torus
        float point[3] = {R2 + R1 * cos(theta), R1 * sin(theta), 0};
        float pointbuf[3][3];
        d_rotate_mat(1, phi, 1, point, pointbuf[0]);
        d_rotate_mat(1, A, 0, pointbuf[0], pointbuf[1]);
        d_rotate_mat(1, B, 2, pointbuf[1], pointbuf[2]);
        //ensure the point is in front of the viewer
        pointbuf[2][2] += DIST;

        point_t temp;
        temp.z_inv = 1 / pointbuf[2][2];
        temp.xp = (int)(COL / 2 + pointbuf[2][0] * ZPRIMEX * temp.z_inv) + offsetx;
        temp.yp = (int)(ROW / 2 - pointbuf[2][1] * ZPRIMEY * temp.z_inv) + offsety;
        
        //accommodate if projection is out-of-bounds
        char invalid_pos =
            temp.xp < 0
            || temp.xp > COL - 1
            || temp.yp < 0
            || temp.yp > ROW - 1;
        if (invalid_pos)
        {
            temp.xp = temp.yp = temp.z_inv = 0;
        }

        //perform the same rotations with unit circle
        //to find surface normal
        float norm[3] = {cos(theta), sin(theta), 0};
        float normbuf[3][3];
        d_rotate_mat(1, phi, 1, norm, normbuf[0]);
        d_rotate_mat(1, A, 0, normbuf[0], normbuf[1]);
        d_rotate_mat(1, B, 2, normbuf[1], normbuf[2]);
        //dot product of normal vector and light source
        //1 parallel, -1 anti-parallel, 0 perpendicular
        float light[3];
        light[0] = 0;
        light[1] = 1 / sqrt(2.0f);
        light[2] = -1 / sqrt(2.0f);
        float light_transpose[3];
        d_trans_mat(1, 3, light, light_transpose);
        float result[1];
        d_mult_mat(1, 3, 3, 1, normbuf[2], light_transpose, result);

        temp.lum = result[0];

        //capture the results for this point (for this frame) in
        //the synchronized array
        points[gindex] = temp;
    }
}

extern "C" void cuda_render_frame(render_args *input)
{

    point_t *d_points;
    hipMalloc(&d_points, sizeof(point_t) * input->img.outer * input->img.inner);
    
    dim3 dimBlock(32, 4);

    dim3 dimGrid((int) ceil(input->img.inner * 1.0f / dimBlock.x), (int) ceil(input->img.outer * 1.0f / dimBlock.y));    
    
  
    hipEvent_t start, stop;
  
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    render_frame<<<dimGrid,dimBlock>>>(input->A, input->B,input->offsetx,input->offsety, input->img.outer, input->img.inner, d_points);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&input->last_frame_time, start, stop);

    
    hipMemcpy(input->img.points, d_points, sizeof(point_t) * input->img.outer * input->img.inner, hipMemcpyDeviceToHost);

    hipFree(d_points);
 
}

extern "C" void cuda_device_reset()
{
    hipDeviceReset();
}