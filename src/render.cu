#include "hip/hip_runtime.h"
#include "params.h"

enum rotationAxis {aX = 0, aY = 1, aZ = 2};

__device__ void d_mult_mat(
        size_t      m1,
        size_t      n1,
        size_t      m2,
        size_t      n2,
        const float *a,
        const float *b,
        float       *buf)
{
    for (size_t i = 0; i < m1; ++i)
        for (size_t j = 0; j < n2; ++j)
        {
            float sum = 0;
            for (size_t k = 0; k < m2; ++k)
                sum += a[i * n1 + k] * b[k * n2 + j];
            buf[i * n2 + j] = sum;
        }
}

__device__ void d_rotate_mat(
        size_t            m1,
        float             phi,
        enum rotationAxis axis,
        const float       *a,
        float             *buf)
{
    switch (axis)
    {
        case aX:
            {
                float R_x[9] =
                {
                    1, 0,        0,
                    0, cos(phi), -1 * sin(phi),
                    0, sin(phi), cos(phi)
                };
                d_mult_mat(m1, 3, 3, 3, a, R_x, buf);
                break;
            }
        case aY:
            {
                float R_y[9] =
                {
                    cos(phi),      0, sin(phi),
                    0,             1, 0,
                    -1 * sin(phi), 0, cos(phi)
                };
                d_mult_mat(m1, 3, 3, 3, a, R_y, buf);
                break;
            }
        case aZ:
            {
                float R_z[9] =
                {
                    cos(phi), -1 * sin(phi), 0,
                    sin(phi), cos(phi),      0,
                    0,        0,             1
                };
                d_mult_mat(m1, 3, 3, 3, a, R_z, buf);
                break;
            }
    }
}

__device__ void d_trans_mat(
        size_t      m,
        size_t      n,
        const float *a,
        float       *buf)
{
    for (size_t i = 0; i < m * n; ++i)
    {
        size_t x = i / m;
        size_t y = i % m;
        buf[i] = a[n * y + x];
    }
}

__global__ void render_frame(
        float   A,
        float   B,
        int     offsetx,
        int     offsety,
        float   lightx,
        float   lighty,
        float   lightz,
        size_t  outer,
        size_t  inner,
        point_t *points)
{
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < outer && col < inner)
    {
        size_t gindex = row * inner + col;
        float  theta  = (float) row * THETA_INC;
        float  phi    = (float) col * PHI_INC;

        //rotate a point of the torus
        float point[3] = {R2 + R1 * cos(theta), R1 * sin(theta), 0};
        float pointbuf[3][3];
        d_rotate_mat(1, phi, aY, point, pointbuf[0]);
        d_rotate_mat(1, A, aX, pointbuf[0], pointbuf[1]);
        d_rotate_mat(1, B, aZ, pointbuf[1], pointbuf[2]);
        //ensure the point is in front of the viewer
        pointbuf[2][2] += DIST;

        point_t temp;
        temp.z_inv = 1 / pointbuf[2][2];
        temp.xp = (int)(COL / 2 + pointbuf[2][0] * ZPRIMEX * temp.z_inv) + offsetx;
        temp.yp = (int)(ROW / 2 - pointbuf[2][1] * ZPRIMEY * temp.z_inv) + offsety;

        //handle case when projection is out-of-bounds
        unsigned char invalid_pos = temp.xp < 0
            || temp.xp > COL - 1
            || temp.yp < 0
            || temp.yp > ROW - 1;
        if (invalid_pos)
        {
            temp.xp = temp.yp = temp.z_inv = 0;
        }

        //perform the same rotations with unit circle
        //to find surface normal
        float norm[3] = {cos(theta), sin(theta), 0};
        float normbuf[3][3];
        d_rotate_mat(1, phi, aY, norm, normbuf[0]);
        d_rotate_mat(1, A, aX, normbuf[0], normbuf[1]);
        d_rotate_mat(1, B, aZ, normbuf[1], normbuf[2]);
        //dot product of surface normal and light source
        //1 parallel, -1 anti-parallel, 0 perpendicular
        float light_src[3] = {lightx, lighty, lightz}, light_src_transpose[3], result[1];
        d_trans_mat(1, 3, light_src, light_src_transpose);
        d_mult_mat(1, 3, 3, 1, normbuf[2], light_src_transpose, result);
        temp.lum = result[0];

        //capture the results for this point (for this frame)
        points[gindex] = temp;
    }
}

extern "C" void cuda_render_frame(render_args *r_args)
{
    point_t *d_points;
    size_t points_arr_size = sizeof(point_t) * r_args->outer * r_args->inner;
    hipMalloc(&d_points, points_arr_size);

    dim3 dimBlock(32, 4);
    dim3 dimGrid((int) ceil(r_args->inner * 1.0f / dimBlock.x),
            (int) ceil(r_args->outer * 1.0f / dimBlock.y));
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    render_frame<<<dimGrid,dimBlock>>>(
            r_args->A,
            r_args->B,
            r_args->offsetx,
            r_args->offsety,
            r_args->light_src[0][0],
            r_args->light_src[0][1],
            r_args->light_src[0][2],
            r_args->outer,
            r_args->inner,
            d_points);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&r_args->frame_time, start, stop);

    hipMemcpy(r_args->points, d_points, points_arr_size, hipMemcpyDeviceToHost);
    hipFree(d_points);
}

extern "C" void cuda_device_reset()
{
    hipDeviceReset();
}
